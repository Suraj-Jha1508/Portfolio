#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
//#include <hip/hip_runtime_api.h>
#include "dp.h"

int main(int argc ,char* argv[]) {
	
	FILE *matrix_data;
	FILE *w;
	size_t size;
	size_t sizew;
	
	/* Initialize rows, cols, CUDA devices and threads from the user */
	unsigned int rows=atoi(argv[3]);
	unsigned int cols=atoi(argv[4]);					  
	int CUDA_DEVICE = atoi(argv[5]);
	int THREADS = atoi(argv[6]);
	
	printf("Rows= %d\n,Cols = %d\n,CUDA_DEVICE= %d\n, THREADS =%d \n",rows,cols,CUDA_DEVICE,THREADS);
	hipError_t err = hipSetDevice(CUDA_DEVICE);

	if(err != hipSuccess) { printf("Error setting CUDA DEVICE\n"); exit(EXIT_FAILURE); }

	/*Host variable declaration */

	//int THREADS = 32;				
	int BLOCKS;
	float* host_results = (float*) malloc(rows * sizeof(float)); 
	struct timeval starttime, endtime;
	clock_t start, end;
	float seconds = 0;
	unsigned int jobs; 
	unsigned long i;

	/*Kernel variable declaration */
	
	float  *d_dataT;
	float *d_dataV;
	float *results;
        //size_t len = 0;
	float arr[rows][cols];
	float var ;
	int vrow =1;

	start = clock();

	/* Validation to check if the data file is readable */
	
	matrix_data = fopen(argv[1], "r");
	w = fopen(argv[2],"r");
	
	if (matrix_data == NULL)
	{
    		printf("Cannot Open the data ");
		return 0;
	}
	if (w == NULL)
	{
    		printf("Cannot Open the vector");
		return 0;
	}
	
	size = (size_t)((size_t)rows * (size_t)cols);
	sizew = (size_t)((size_t)vrow*(size_t)cols);

	printf("Size of the data = %lu\n",size);
	printf("Size of the vector = %lu\n",sizew);

	fflush(stdout);

	float *dataT = (float*)malloc((size)*sizeof(float));
	float *dataV = (float*)malloc((sizew)*sizeof(float));

	if(dataT == NULL) {
	        printf("ERROR: Memory for data not allocated.\n");
	}
	if(dataV == NULL) {
	        printf("ERROR: Memory for vector not allocated.\n");
	}
	
        gettimeofday(&starttime, NULL);
	int j = 0;

    /* Transfer the Data from the file to CPU Memory */
	

        for (i =0; i< rows;i++){
		for(j=0; j<cols ; j++){
			fscanf(matrix_data,"%f",&var);
                        arr[i][j]=var;
		}
	}
	for (i =0;i<cols;i++){
		for(j= 0; j<rows; j++){
			dataT[rows*i+j]= arr[j][i];
	}
	}		

		for (j=0;j<cols;j++){
			fscanf(w,"%f",&dataV[j]);
		}
   
	fclose(matrix_data);
	fclose(w);
		printf("Read Data\n");
        fflush(stdout);

        gettimeofday(&endtime, NULL);
        seconds+=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);

        printf("time to read data = %f\n", seconds);

	/* Allocate the Memory in the GPU for data */

        gettimeofday(&starttime, NULL);
	err = hipMalloc((float**) &d_dataT, (size_t) size * (size_t) sizeof(float));
	if(err != hipSuccess) { printf("Error mallocing data on GPU device\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time for cudamalloc for data =%f\n", seconds);

        gettimeofday(&starttime, NULL);


	/* Allocate the memory in the GPU for vector */
	
        err = hipMalloc((float**) &d_dataV, sizew * sizeof(float));
       if(err != hipSuccess) { printf("Error mallocing data on GPU device\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
        printf("time for cudamalloc for vector =%f\n", seconds);

        gettimeofday(&starttime, NULL);
	
	err = hipMalloc((float**) &results, rows * sizeof(float) );
	if(err != hipSuccess) { printf("Error mallocing results on GPU device\n"); }
        gettimeofday(&endtime, NULL); 
seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time for cudamalloc for result =%f\n", seconds);

	/*Copy the data to GPU */
	
	
        gettimeofday(&starttime, NULL);
	err = hipMemcpy(d_dataT, dataT, (size_t)size *sizeof(float), hipMemcpyHostToDevice);
	if(err != hipSuccess) { printf("Error copying data to GPU\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time to copy  data to GPU=%f\n", seconds);

	
	gettimeofday(&starttime, NULL);
        err = hipMemcpy(d_dataV, dataV, sizew*sizeof(float), hipMemcpyHostToDevice);
        if(err != hipSuccess) { printf("Error copying data to GPU\n"); }
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
        printf("time to copy vector data to GPU=%f\n", seconds);

	jobs = rows;
	BLOCKS = (jobs + THREADS - 1)/THREADS;

        gettimeofday(&starttime, NULL);

	/* Calling the kernel function */
	
	kernel<<<BLOCKS,THREADS>>>(rows,cols,d_dataT,	d_dataV, results);
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
	printf("time for kernel=%f\n", seconds);
		
	/* Copy the results back in host */
	
	hipMemcpy(host_results,results,rows * sizeof(float),hipMemcpyDeviceToHost);
	
	printf("Output of dot product is \n");
	printf("\n");
	
	for(int k = 0; k < jobs; k++) {
		printf("%f ", host_results[k]);
	}
	printf("\n");

	hipFree( d_dataT );
	hipFree( results );

	end = clock();
	seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Total time = %f\n", seconds);

	return 0;

}