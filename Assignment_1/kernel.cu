#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "dp.h"

__global__ void kernel(unsigned int rows, unsigned int cols , float* ddata,float* vdata ,float *results){
	
	int i;
        float sum =0;
	int tid  = threadIdx.x + blockIdx.x * blockDim.x;
	
	for(i =0; i<cols ;i++ )
	{
		sum+= ddata[i*rows+tid]*vdata[i];		
	}
	
	results[tid] = sum;
	
}
